#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h> 
#include <string.h>

// Parallel Computing Lab 3
// Abhi Dankar
// Professor Zahran
// All experiments run on cuda1

unsigned int getmax(unsigned int *, unsigned int);
unsigned int getmaxSEQ(unsigned int *, unsigned int);
void printArr(unsigned int pr[], unsigned int box);

#define THREADS_PER_BLOCK 1024

__global__ void getMaxCu(unsigned int* arrIn, unsigned int size, unsigned int* res) {
  unsigned int i = 0;
  __shared__ unsigned int arr[THREADS_PER_BLOCK];
  i =  threadIdx.x + (blockIdx.x * blockDim.x);
  unsigned int gb = gridDim.x*blockDim.x;
  unsigned int o = 0;

  //find max within block
  unsigned int temp = 0;
  while(i + o < size){
    temp = max(temp, arrIn[i + o]);
    o += gb;
  }
  arr[threadIdx.x] = temp;
  __syncthreads();

  //across blocks
  for (unsigned int y = (blockDim.x/2); y > 0; y = y/2){
      if ((threadIdx.x < y)){
        arr[threadIdx.x] = max(arr[threadIdx.x], arr[threadIdx.x + y]);
      }
      __syncthreads();
  }
  //get the max to return from 0 thread
  if (threadIdx.x == 0)
    res[blockIdx.x] = max(res[blockIdx.x],arr[0]);
}



int main(int argc, char *argv[]) {
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array
    
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }    

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
       numbers[i] = rand()  % size;  

    printf(" The maximum number in the array is: %u\n", 
           getmax(numbers, size));

    free(numbers);
    exit(0);
}

void printArr(unsigned int pr[], unsigned int box){
  for (unsigned int i = 0; i < box; i++){
    printf("%u ", pr[i]);
  } 
}

//sequential getMax for checking
unsigned int getmaxSEQ(unsigned int num[], unsigned int size) {
  unsigned int i;
  unsigned int max = num[0];
  for(i = 1; i < size; i++){
    if(num[i] > max){
      max = num[i];
   }
  }
  return( max );
}
/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/
unsigned int getmax(unsigned int num[], unsigned int size) {
  unsigned int i;
  unsigned int * copy;
  unsigned int * ans; 
  unsigned int * output;
  unsigned int * newArr;
  unsigned int resize;
  //in case array doesn't fill up the block
  if (size % THREADS_PER_BLOCK != 0){
    resize = (size/THREADS_PER_BLOCK+1)*THREADS_PER_BLOCK;
  } else {
    resize = size;
  }
  //create new array with 0 values for unfilled block
  newArr = (unsigned int *) malloc(sizeof(unsigned int) * resize);
  for (i = 0; i < resize; i++){
    if (i < size){
      newArr[i] = num[i];
    } else {
      newArr[i] = 0;
    }
  }

  //how many blocks
  unsigned int blocks = (resize/THREADS_PER_BLOCK);
  hipMalloc((void **) &copy, sizeof(unsigned int)*resize);
  hipMemcpy((void *)copy, (void *) newArr, resize*sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMalloc((void **) &output, sizeof(unsigned int)*blocks);
  ans = (unsigned int*) malloc(sizeof(unsigned int) * blocks);

  do {
    blocks = ceil((float)(resize)/(float)THREADS_PER_BLOCK);
    getMaxCu<<<blocks, THREADS_PER_BLOCK>>>(copy, resize, output);
    resize = blocks;
    copy = output;
  } while (blocks > 1);

  hipMemcpy((void *)ans, (void *)output, blocks * sizeof(unsigned int),hipMemcpyDeviceToHost);
  unsigned int ret = ans[0];
  hipFree(output);
  hipFree(copy);
  free(newArr);
  free(ans);
  return(ret);
}


